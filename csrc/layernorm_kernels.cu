#include "hip/hip_runtime.h"
#include "type_convert.cuh"
#include "dispatch_utils.h"

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/hipcub.hpp>
#endif

#ifdef USE_ROCM
  #include "quantization/fp8/amd/quant_utils.cuh"
#else
  #include "quantization/fp8/nvidia/quant_utils.cuh"
#endif

#if defined(__HIPCC__) &&                                                 \
    (defined(__gfx90a__) || defined(__gfx940__) || defined(__gfx941__) || \
     defined(__gfx942__) || defined(__gfx1100__))
  #define __HIP__MI300_MI250_Navi31__
#endif

namespace vllm {

template <typename scalar_t>
struct __align__(16) vec8_t {
  scalar_t x, y, z, w, u, v, s, t;

  __device__ vec8_t() : x(0), y(0), z(0), w(0), u(0), v(0), s(0), t(0) {}
  __device__ vec8_t(scalar_t x, scalar_t y, scalar_t z, scalar_t w, scalar_t u,
                    scalar_t v, scalar_t s, scalar_t t)
      : x(x), y(y), z(z), w(w), u(u), v(v), s(s), t(t) {}

  __device__ vec8_t operator*(const vec8_t& other) const {
    return vec8_t(x * other.x, y * other.y, z * other.z, w * other.w,
                  u * other.u, v * other.v, s * other.s, t * other.t);
  }

  __device__ vec8_t operator*(const float& scale) const {
    return vec8_t(x * scale, y * scale, z * scale, w * scale, u * scale,
                  v * scale, s * scale, t * scale);
  }

  __device__ vec8_t operator+(const vec8_t& other) const {
    return vec8_t(x + other.x, y + other.y, z + other.z, w + other.w,
                  u + other.u, v + other.v, s + other.s, t + other.t);
  }

  __device__ void operator+=(const vec8_t& other) {
    x += other.x;
    y += other.y;
    z += other.z;
    w += other.w;
    u += other.u;
    v += other.v;
    s += other.s;
    t += other.t;
  }

  __device__ scalar_t sum() const { return x + y + z + w + u + v + s + t; }
};

#ifdef __HIP__MI300_MI250_Navi31__

// TODO(woosuk): Further optimize this kernel.
template <typename scalar_t>
__global__ void rms_norm_kernel(
    scalar_t* __restrict__ out,           // [..., hidden_size]
    const scalar_t* __restrict__ input,   // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;

  vec8_t<scalar_t> v8_variance = {0, 0, 0, 0, 0, 0, 0, 0};

  vec8_t<scalar_t>* vectorized_out = reinterpret_cast<vec8_t<scalar_t>*>(out);
  vec8_t<scalar_t> const* vectorized_in =
      reinterpret_cast<vec8_t<scalar_t> const*>(input);
  vec8_t<scalar_t> const* vectorized_weight =
      reinterpret_cast<vec8_t<scalar_t> const*>(weight);
  const int vec_hidden_size = hidden_size >> 3;

  // Compute variance. Be careful, hidden_size should multiple of 4.
  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    vec8_t<scalar_t> x = vectorized_in[blockIdx.x * vec_hidden_size + idx];
    v8_variance += x * x;
  }
  float v8_variance_sum = v8_variance.sum();

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  float variance =
      BlockReduce(reduceStore).Reduce(v8_variance_sum, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    vec8_t<scalar_t> v8_in = vectorized_in[blockIdx.x * vec_hidden_size + idx];
    vec8_t<scalar_t> v8_w = vectorized_weight[idx];
    vectorized_out[blockIdx.x * vec_hidden_size + idx] =
        v8_in * s_variance * v8_w;
  }
}

#else

// TODO(maleksan): Investigate why vectorization doesn't work for Navi.
template <typename scalar_t>
__global__ void rms_norm_kernel(
    scalar_t* __restrict__ out,           // [..., hidden_size]
    const scalar_t* __restrict__ input,   // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

#endif

/* Function specialization in the case of FP16/BF16 tensors.
   Additional optimizations we can make in this case are
   packed and vectorized operations, which help with the
   memory latency bottleneck. */
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width > 0) && _typeConvert<scalar_t>::exists>
fused_add_rms_norm_kernel(
    scalar_t* __restrict__ input,         // [..., hidden_size]
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  // Sanity checks on our vector struct and type-punned pointer arithmetic
  static_assert(std::is_pod_v<_f16Vec<scalar_t, width>>);
  static_assert(sizeof(_f16Vec<scalar_t, width>) == sizeof(scalar_t) * width);

  const int vec_hidden_size = hidden_size / width;
  __shared__ float s_variance;
  float variance = 0.0f;
  /* These and the argument pointers are all declared `restrict` as they are
     not aliased in practice. Argument pointers should not be dereferenced
     in this kernel as that would be undefined behavior */
  auto* __restrict__ input_v =
      reinterpret_cast<_f16Vec<scalar_t, width>*>(input);
  auto* __restrict__ residual_v =
      reinterpret_cast<_f16Vec<scalar_t, width>*>(residual);
  auto* __restrict__ weight_v =
      reinterpret_cast<const _f16Vec<scalar_t, width>*>(weight);

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _f16Vec<scalar_t, width> temp = input_v[id];
    temp += residual_v[id];
    variance += temp.sum_squares();
    residual_v[id] = temp;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _f16Vec<scalar_t, width> temp = residual_v[id];
    temp *= s_variance;
    temp *= weight_v[idx];
    input_v[id] = temp;
  }
}

/* Generic fused_add_rms_norm_kernel
   The width field is not used here but necessary for other specializations.
 */
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width == 0) || !_typeConvert<scalar_t>::exists>
fused_add_rms_norm_kernel(
    scalar_t* __restrict__ input,         // [..., hidden_size]
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    scalar_t z = input[blockIdx.x * hidden_size + idx];
    z += residual[blockIdx.x * hidden_size + idx];
    float x = (float)z;
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = z;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)residual[blockIdx.x * hidden_size + idx];
    input[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

/* Function specialization in the case of FP16/BF16 tensors.
   Additional optimizations we can make in this case are
   packed and vectorized operations, which help with the
   memory latency bottleneck. */

template <>
struct Vec<c10::Float8_e4m3fnuz, 8> {
  using Type = uint2;
};

template <>
struct Vec<c10::Half, 8> {
  using Type = uint4;
};

template <>
struct Vec<c10::BFloat16, 8> {
  using Type = bf16_8_t;
};

}  // namespace vllm

void rms_norm(torch::Tensor& out,     // [..., hidden_size]
              torch::Tensor& input,   // [..., hidden_size]
              torch::Tensor& weight,  // [hidden_size]
              double epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    vllm::rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
  });
}

#define LAUNCH_FUSED_ADD_RMS_NORM(width)                                       \
  VLLM_DISPATCH_FLOATING_TYPES(                                                \
      input.scalar_type(), "fused_add_rms_norm_kernel", [&] {                  \
        vllm::fused_add_rms_norm_kernel<scalar_t, width>                       \
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),           \
                                         residual.data_ptr<scalar_t>(),        \
                                         weight.data_ptr<scalar_t>(), epsilon, \
                                         num_tokens, hidden_size);             \
      });

void fused_add_rms_norm(torch::Tensor& input,     // [..., hidden_size]
                        torch::Tensor& residual,  // [..., hidden_size]
                        torch::Tensor& weight,    // [hidden_size]
                        double epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  /* This kernel is memory-latency bound in many scenarios.
     When num_tokens is large, a smaller block size allows
     for increased block occupancy on CUs and better latency
     hiding on global mem ops. */
  const int max_block_size = (num_tokens < 256) ? 1024 : 256;
  dim3 block(std::min(hidden_size, max_block_size));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  /*If the tensor types are FP16/BF16, try to use the optimized kernel
    with packed + vectorized ops.
    Max optimization is achieved with a width-8 vector of FP16/BF16s
    since we can load at most 128 bits at once in a global memory op.
    However, this requires each tensor's data to be aligned to 16
    bytes.
   */
  auto inp_ptr = reinterpret_cast<std::uintptr_t>(input.data_ptr());
  auto res_ptr = reinterpret_cast<std::uintptr_t>(residual.data_ptr());
  auto wt_ptr = reinterpret_cast<std::uintptr_t>(weight.data_ptr());
  bool ptrs_are_aligned =
      inp_ptr % 16 == 0 && res_ptr % 16 == 0 && wt_ptr % 16 == 0;
  if (ptrs_are_aligned && hidden_size % 8 == 0) {
    LAUNCH_FUSED_ADD_RMS_NORM(8);
  } else {
    LAUNCH_FUSED_ADD_RMS_NORM(0);
  }
}
